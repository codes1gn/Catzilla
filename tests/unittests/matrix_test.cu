#define CATCH_CONFIG_MAIN
#include <catch2/catch_test_macros.hpp>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "matrix_utils.h"

using namespace catz;

TEST_CASE("matrix construction", "[matrix][construction]") {
  const int M_TILE = 32;
  const int K_TILE = 32;
  MAKE_SHARED(lhs_shared_mat, M_TILE, K_TILE, float);

  static_assert(lhs_shape_mat.shape.first == 32,
                "matrix.shape.first not available at compile-time");
  // static_assert(lhs_shape_mat.shape.second == 2,
  //               "matrix.shape.first not available at compile-time");
}
