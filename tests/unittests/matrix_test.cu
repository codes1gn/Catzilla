#define CATCH_CONFIG_MAIN
#include <catch2/catch_test_macros.hpp>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "matrix.h"

using namespace catz;

TEST_CASE("matrix construction", "[matrix][construction]") {
  const int M_TILE = 32;
  const int K_TILE = 32;
  auto _shape = CoordNightly<M_TILE, K_TILE>();
  auto _stride = CoordNightly<K_TILE, 1>();
  float lhs_data[M_TILE*K_TILE] = {0.0};
  // MatrixNightly<float, CoordNightly> lhs_mat(_shape);
  auto lhs_mat = make_matrix(lhs_data, _shape, _stride);

  static_assert(lhs_mat.shape.first == 32,
                "matrix.shape.first not available at compile-time");
  static_assert(lhs_mat.shape.second == 32,
                "matrix.shape.first not available at compile-time");

  static_assert(lhs_mat.stride.first == 32,
                "matrix.stride.first not available at compile-time");
  static_assert(lhs_mat.stride.second == 1,
                "matrix.stride.first not available at compile-time");
}
