#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "verify.h"
#include "hipDNN.h"

int main(int argc, char **argv)
{
    unsigned int n = atoi(argv[1]);
    unsigned int c = atoi(argv[2]);
    unsigned int h = atoi(argv[3]);
    unsigned int w = atoi(argv[4]);
    unsigned int k = atoi(argv[5]);
    unsigned int r = atoi(argv[6]);
    unsigned int s = atoi(argv[7]);
    unsigned int u = atoi(argv[8]);
    unsigned int v = atoi(argv[9]);
    unsigned int p = atoi(argv[10]);
    unsigned int q = atoi(argv[11]);

    int outh = (h - r + 2 * p) / u + 1;
    int outw = (w - s + 2 * q) / v + 1;
    double M = k;
    double N = n * outh * outw;
    double K = c * r * s;
    double temp = n * outh * outw * 1e-9f;
    double flopsPerConv = temp * M * K * 2.0;
    float *input = (float *)malloc(n * c * h * w * sizeof(float));
    float *weight = (float *)malloc(k * c * r * s * sizeof(float));
    float *output = (float *)malloc(n * k * outh * outw * sizeof(float));
    float *output_host = (float *)malloc(n * k * outh * outw * sizeof(float));

    float *input_device, *weight_device, *output_device;
    hipMalloc((void **)&input_device, n * c * h * w * sizeof(float));
    hipMalloc((void **)&weight_device, k * c * r * s * sizeof(float));
    hipMalloc((void **)&output_device, n * k * outh * outw * sizeof(float));

    for (int i = 0; i < n * c * h * w; i++)
    {
        input[i] = (rand() % 255) / 255.0;
    }

    for (int i = 0; i < k * c * r * s; i++)
    {
        weight[i] = (rand() % 255) / 255.0;
    }

    for (int i = 0; i < n * k * outh * outw; i++)
    {
        output[i] = 0.0;
        output_host[i] = 0.0;
    }

    hipMemcpy(input_device, input, n * c * h * w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weight_device, weight, k * c * r * s * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(output_device, output, n * k * outh * outw * sizeof(float), hipMemcpyHostToDevice);


    hipdnnStatus_t status;
    hipdnnHandle_t handle;
    status = hipdnnCreate(&handle);
    if (status != HIPDNN_STATUS_SUCCESS)
        printf("hipdnnCreate failed\n");
    hipdnnTensorDescriptor_t input_descriptor;
    status = hipdnnCreateTensorDescriptor(&input_descriptor);
    if (status != HIPDNN_STATUS_SUCCESS)
        printf("hipdnnCreateTensorDescriptor input_descriptor failed\n");
    status = hipdnnSetTensor4dDescriptor(input_descriptor,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/n,
            /*channels=*/c,
            /*image_height=*/h,
            /*image_width=*/w);
    if (status != HIPDNN_STATUS_SUCCESS)
        printf("hipdnnSetTensor4dDescriptor input_descriptor failed\n");
    hipdnnFilterDescriptor_t kernel_descriptor;
    status = hipdnnCreateFilterDescriptor(&kernel_descriptor);
    if (status != HIPDNN_STATUS_SUCCESS)
        printf("hipdnnCreateTensorDescriptor kernel_descriptor failed\n");
    status = hipdnnSetFilter4dDescriptor(kernel_descriptor,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*out_channels=*/k,
            /*in_channels=*/c,
            /*kernel_height=*/r,
            /*kernel_width=*/s);
    if (status != HIPDNN_STATUS_SUCCESS)
        printf("hipdnnSetTensor4dDescriptor kernel_descriptor failed\n");
    hipdnnConvolutionDescriptor_t convolution_descriptor;
    status = hipdnnCreateConvolutionDescriptor(&convolution_descriptor);
    if (status != HIPDNN_STATUS_SUCCESS)
        printf("hipdnnCreateConvolutionDescriptor convolution_descriptor failed\n");
    status = hipdnnSetConvolution2dDescriptor(convolution_descriptor,
            /*pad_height=*/p,
            /*pad_width=*/q,
            /*vertical_stride=*/u,
            /*horizontal_stride=*/v,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/HIPDNN_CROSS_CORRELATION, // how to compute
            /*computeType=*/HIPDNN_DATA_FLOAT);
    if (status != HIPDNN_STATUS_SUCCESS)
        printf("hipdnnSetTensor4dDescriptor convolution_descriptor failed\n");

    hipdnnMathType_t mathType=HIPDNN_TENSOR_OP_MATH;  //choose FMA or Tensor math

    // HIPDNN_DEFAULT_MATH                    = 0,
    // HIPDNN_TENSOR_OP_MATH                  = 1,
    // CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION = 2,
    // CUDNN_FMA_MATH                        = 3,

    status = hipdnnSetConvolutionMathType(convolution_descriptor,mathType);

    if (status != HIPDNN_STATUS_SUCCESS)
        printf("hipdnnSetConvolutionMathType failed\n");


    // create output descriptor
    hipdnnTensorDescriptor_t output_descriptor;
    status = hipdnnCreateTensorDescriptor(&output_descriptor);
    if (status != HIPDNN_STATUS_SUCCESS)
        printf("hipdnnCreateTensorDescriptor output_descriptor failed\n");
    status = hipdnnSetTensor4dDescriptor(output_descriptor,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/n,
            /*channels=*/k,
            /*image_height=*/outh,
            /*image_width=*/outw);
    if (status != HIPDNN_STATUS_SUCCESS)
        printf("hipdnnSetTensor4dDescriptor output_descriptor failed\n");

    hipdnnConvolutionFwdAlgoPerf_t perfResults[9];
    int returnedAlgoCount;
    status = hipdnnFindConvolutionForwardAlgorithm(handle,
                                                  input_descriptor,
                                                  kernel_descriptor,
                                                  convolution_descriptor,
                                                  output_descriptor,
                                                  9,
                                                  &returnedAlgoCount,
                                                  perfResults);
    if (status != HIPDNN_STATUS_SUCCESS)
        printf("hipdnnFindConvolutionForwardAlgorithm failed\n");

    // print all available convolution forward algorithm , ordered by time
    // for (int i = 0; i < 9; i++)
    // {
    //     printf("Algorithm %d: %d, time: %f\n", i, perfResults[i].algo, perfResults[i].time);
    // }

    // cuDNN all convolution forward algorithm

    // HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM         = 0
    // HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM = 1
    // HIPDNN_CONVOLUTION_FWD_ALGO_GEMM                  = 2
    // HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT                = 3
    // HIPDNN_CONVOLUTION_FWD_ALGO_FFT                   = 4
    // HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING            = 5
    // HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD              = 6
    // HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED     = 7
    // HIPDNN_CONVOLUTION_FWD_ALGO_COUNT                 = 8

    hipdnnConvolutionFwdAlgo_t convolution_algorithm = (hipdnnConvolutionFwdAlgo_t)0;  //choose implicit gemm

    int size;
    status = hipdnnGetConvolutionForwardWorkspaceSize(handle,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     convolution_descriptor,
                                                     output_descriptor,
                                                     convolution_algorithm,
                                                     (size_t *)&(size));
    // printf("Workspace size: %zu bytes\n", size);

    if (status != HIPDNN_STATUS_SUCCESS)
        printf("hipdnnGetConvolutionForwardWorkspaceSize failed\n");
    float *extra;
    hipMalloc((void **)&extra, size);

    float alpha = 1.0, beta = 0.0;

    status = hipdnnConvolutionForward(handle, &alpha,
                                     input_descriptor, input_device, kernel_descriptor, weight_device,
                                     convolution_descriptor, convolution_algorithm,
                                     extra, size, &beta,
                                     output_descriptor, output_device);

    if (status != HIPDNN_STATUS_SUCCESS)
        printf("Not Successed!\n");
    hipMemcpy(output_host, output_device, n * k * outh * outw * sizeof(float), hipMemcpyDeviceToHost);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int warmup = 10;
    for (int i = 0; i < warmup; i++)
    {
        hipdnnConvolutionForward(handle, &alpha,
                                input_descriptor, input_device, kernel_descriptor, weight_device,
                                convolution_descriptor, convolution_algorithm,
                                extra, size, &beta,
                                output_descriptor, output_device);
    }
    hipDeviceSynchronize();
    printf("warmup finished\n");

    hipEventRecord(start, 0);
    float time_elapsed = 0.0;

    int iternum = 10;
    for (int i = 0; i < iternum; i++)
    {
        hipdnnConvolutionForward(handle, &alpha,
                                input_descriptor, input_device, kernel_descriptor, weight_device,
                                convolution_descriptor, convolution_algorithm,
                                extra, size, &beta,
                                output_descriptor, output_device);
    }
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // printf("===================start verfiy===================\n");
    // direct_conv2dcpu(input, weight, output, n, c, h, w, k, r, s, u, v, p, q);

    // int error = 0;
    // for (int i = 0; i < n * k * outh * outw; i++)
    // {
    //     if (abs(output_host[i] - output[i]) > getPrecision(output[i]))
    //     {
    //         printf("error, postion:%d, gpuvalue:%f, cpuvalue:%f\n", i, output_host[i], output[i]);
    //         error++;
    //         break;
    //     }
    // }
    // printf("================finish,error:%d=========================\n", error);

    float timePerConv = time_elapsed / iternum;
    double gflops = flopsPerConv / (timePerConv / 1000.0f);
    printf("%2d %2d %2d %2d %d %d %2d\n", n, h, w, c, r, s, k);
    printf("time: %f ms\n", timePerConv);
    printf("Performance :%f GFlops\n",  gflops);

    hipFree(input_device);
    hipFree(weight_device);
    hipFree(output_device);

    free(input);
    free(weight);
    free(output);
    free(output_host);

    return 0;
}
