#include "hip/hip_runtime.h"
#include <stdio.h>
#include "sputnik/cuda_utils.h"
#include "sputnik/depthwise/cuda_depthwise.h"

using namespace sputnik;

int main(int argc, char **argv)
{
    unsigned int n = atoi(argv[1]);
    unsigned int c = atoi(argv[2]);
    unsigned int h = atoi(argv[3]);
    unsigned int w = atoi(argv[4]);
    unsigned int k = atoi(argv[5]);
    unsigned int r = atoi(argv[6]);
    unsigned int s = atoi(argv[7]);
    unsigned int u = atoi(argv[8]);
    unsigned int v = atoi(argv[9]);
    unsigned int p = atoi(argv[10]);
    unsigned int q = atoi(argv[11]);

    int outh = (h - r + 2 * p) / u + 1;
    int outw = (w - s + 2 * q) / v + 1;
    double M = k;
    double N = n * outh * outw;
    double K = c * r * s;
    double temp = n * outh * outw * 1e-9f;
    // double flopsPerConv = temp * M * K * 2.0;
    double flopsPerConv = temp * M * 2.0;
    float *input = (float *)malloc(n * c * h * w * sizeof(float));
    float *weight = (float *)malloc(k * c * r * s * sizeof(float));
    float *output = (float *)malloc(n * k * outh * outw * sizeof(float));
    float *output_host = (float *)malloc(n * k * outh * outw * sizeof(float));

    float *input_device, *weight_device, *output_device;
    hipMalloc((void **)&input_device, n * c * h * w * sizeof(float));
    hipMalloc((void **)&weight_device, k * c * r * s * sizeof(float));
    hipMalloc((void **)&output_device, n * k * outh * outw * sizeof(float));

    for (int i = 0; i < n * c * h * w; i++)
    {
        input[i] = (rand() % 255) / 255.0;
    }

    for (int i = 0; i < k * c * r * s; i++)
    {
        weight[i] = (rand() % 255) / 255.0;
    }

    for (int i = 0; i < n * k * outh * outw; i++)
    {
        output[i] = 0.0;
        output_host[i] = 0.0;
    }

    hipMemcpy(input_device, input, n * c * h * w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weight_device, weight, k * c * r * s * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(output_device, output, n * k * outh * outw * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int warmup = 10;
    for (int i = 0; i < warmup; i++)
    {
        CudaDepthwise(n, c, h, w, input_device,
                      r, p, u, weight_device,
                      output_device, /*stream=*/0);
    }
    hipDeviceSynchronize();
    printf("warmup finished\n");

    hipEventRecord(start, 0);
    float time_elapsed = 0.0;

    int iternum = 10;
    for (int i = 0; i < iternum; i++)
    {
        CudaDepthwise(n, c, h, w, input_device,
                      r, p, u, weight_device,
                      output_device, /*stream=*/0);
    }
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // printf("===================start verfiy===================\n");
    // direct_conv2dcpu(input, weight, output, n, c, h, w, k, r, s, u, v, p, q);

    // int error = 0;
    // for (int i = 0; i < n * k * outh * outw; i++)
    // {
    //     if (abs(output_host[i] - output[i]) > getPrecision(output[i]))
    //     {
    //         printf("error, postion:%d, gpuvalue:%f, cpuvalue:%f\n", i, output_host[i], output[i]);
    //         error++;
    //         break;
    //     }
    // }
    // printf("================finish,error:%d=========================\n", error);

    float timePerConv = time_elapsed / iternum;
    double gflops = flopsPerConv / (timePerConv / 1000.0f);
    printf("flopsPerConv:%f\n", flopsPerConv);
    printf("%2d %2d %2d %2d %d %d %2d\n", n, h, w, c, r, s, k);
    printf("time: %f ms\n", timePerConv);
    printf("Performance :%f GFlops\n",  gflops);

    hipFree(input_device);
    hipFree(weight_device);
    hipFree(output_device);

    free(input);
    free(weight);
    free(output);
    free(output_host);

    return 0;
}